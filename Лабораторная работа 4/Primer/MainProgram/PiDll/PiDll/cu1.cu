#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string>
using namespace std;

hipError_t piWithCuda(int iter3, double* piCuda);
int showGPU();
//��������� ��������
__global__ void piIter(double* piCuda)
{
	//for (int k = 0; k <= *t - 1; k++)
	//{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
		piCuda[i] = 4.0 / (4.0 * (double)i + 1) - 4.0 / (4.0 * (double)i + 3);
	//}
}
double PiGPU2(int iter2)
{
	/*const int arraySize2 = 300000;
	//
	double c2[arraySize2] = { 0 };*/
	//iter2 = 10000;
	double Pi = 0;
	double *pi = new double[iter2];
	//double pi[1000000] = {};
	hipError_t cudaStatus2 = piWithCuda(iter2, pi);
	for (int i = 0; i < iter2; i++) Pi += pi[i];
	cudaStatus2 = hipDeviceReset();
	return Pi;
}
hipError_t piWithCuda(int iter3, double* piCuda)
{
	//int* dev_iter = 0;
	double* dev_pi = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	int threadsPerBlock = 1024;
	//int size = 100000;
	
	//cudaStatus = hipMalloc((void**)&dev_iter, sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_pi, iter3 * sizeof(double));
	//cudaStatus = hipMemcpy(dev_iter, &iter3, sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_pi, piCuda, iter3 * sizeof(double), hipMemcpyHostToDevice);
	int blocksPerGrid = (iter3 + threadsPerBlock - 1) / threadsPerBlock;
	piIter << <blocksPerGrid, threadsPerBlock >> > (dev_pi);
	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(piCuda, dev_pi, iter3 * sizeof(double), hipMemcpyDeviceToHost);
	//hipFree(dev_iter);
	hipFree(dev_pi);
	return cudaStatus;
}
int showGPU()
{
	int deviceCount;
	hipDeviceProp_t devProp;
	hipGetDeviceCount(&deviceCount);
	string str = "";
	str += "Found devices: " + to_string(deviceCount) + " devices\n";
	for (int device = 0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);
		str += "Device: " + to_string(device);
		str += "\nCompute capability: " + to_string(devProp.major) + "." + to_string(devProp.minor);
		str += "\nName: ";
		str += devProp.name;
		str += "\nTotal Global Memory: " + to_string(devProp.totalGlobalMem);
		str += "\nShared memory per block: " + to_string(devProp.sharedMemPerBlock);
		str += "\nRegisters per block: " + to_string(devProp.regsPerBlock);
		str += "\nWarp size: " + to_string(devProp.warpSize);
		str += "\nMax threads per block: " + to_string(devProp.maxThreadsPerBlock);
		str += "\nTotal constant memory: " + to_string(devProp.totalConstMem);
	}
	/*const char* msg = "Found devices: ";
	const char* msg2 = " Found2 devices: ";
	//char* sum = new char[strlen(msg) + strlen(msg2) + 1];
	//strcpy(sum, msg);
	//strcat(sum, msg2);
	char filename[] = "data.txt";
	// ������ � ����
	FILE* fp = fopen(filename, "w");
	if (fp)
	{
		// ���������� ������
		fputs(msg, fp);
		fclose(fp);
		//printf("File has been written\n");
	}

	delete[] msg;
	delete[] msg2;*/
	//� ������� ���������� file ����� ������������ ������ � �����
	char buf[1024];
	strcpy(buf, str.c_str());
	FILE* file;
	//��������� ��������� ���� � ������� �� ������
	file = fopen("test.txt", "w+t");
	//����� � ����
	fprintf(file, buf);
	//��������� ����
	fclose(file);
	return 0;
	/*string str;
	int deviceCount;
	hipDeviceProp_t devProp;
	hipGetDeviceCount(&deviceCount);
	str += "Found devices: " + to_string(deviceCount) + " devices.\n";
	for (int device = 0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);
		str += "Device: " + to_string(device);
		str += "\nCompute capability: " + to_string(devProp.major) + "." + to_string(devProp.minor);
		str += "\nName ";
		str += devProp.name;
		str += "\nTotal Global Memory " + to_string(devProp.totalGlobalMem);
		str += "\nShared memory per block " + to_string(devProp.sharedMemPerBlock);
		str += "\nRegisters per block " + to_string(devProp.regsPerBlock);
		str += "\nWarp size " + to_string(devProp.warpSize);
		str += "\nMax threads per block " + to_string(devProp.maxThreadsPerBlock);
		str += "\nTotal constant memory " + to_string(devProp.totalConstMem);
	}
	return str;*/
	/*int deviceCount;
	hipDeviceProp_t devProp;
	hipGetDeviceCount(&deviceCount);
	printf("Found %d devices\n", deviceCount);
	for (int device = 0; device < deviceCount; device++)
	{
		hipGetDeviceProperties(&devProp, device);
		printf("Device %d\n", device);
		printf("Compute capability     : %d.%d\n", devProp.major, devProp.minor);
		printf("Name                   : %s\n", devProp.name);
		printf("Total Global Memory    : %d\n", devProp.totalGlobalMem);
		printf("Shared memory per block: %d\n", devProp.sharedMemPerBlock);
		printf("Registers per block    : %d\n", devProp.regsPerBlock);
		printf("Warp size              : %d\n", devProp.warpSize);
		printf("Max threads per block  : %d\n", devProp.maxThreadsPerBlock);
		printf("Total constant memory  : %d\n", devProp.totalConstMem);
		//maxThreadsPerBlock = devProp.maxThreadsPerBlock;
	}*/
	/*return 0;

	// ������ ��� ������
	char* message = str;
	// ���� ��� ������
	char* filename = "data.txt";
	// ������ � ����
	FILE* fp = fopen(filename, "w");
	if (fp)
	{
		// ���������� ������
		fputs(message, fp);
		fclose(fp);
		printf("File has been written\n");
	}*/
}